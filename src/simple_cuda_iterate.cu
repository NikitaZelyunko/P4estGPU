#include "hip/hip_runtime.h"
#include "simple_cuda_iterate.h"

__global__ void
simple_quadrants_iterate(
  p4est_quadrant_t* quadrants, void* quads_data, size_t quad_data_size, p4est_ghost_t* ghost_layer,
  p4est_t* p4est, p4est_topidx_t treeId, 
  void* user_data, cuda_iter_volume_t iter_volume,
  size_t quads_count, size_t quads_per_thread)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  size_t cursor = i * quads_per_thread; 
  if(cursor < quads_count) {
    size_t elems_remaining;
    if(cursor >= quads_count - quads_per_thread) {
      elems_remaining = quads_count - cursor;
    } else {
      elems_remaining = quads_per_thread;
    }
    p4est_quadrant_t *quad_cursor = quadrants + cursor;
    void* quad_data_cursor = quads_data + cursor * quad_data_size;
    for(int j = 0; j < elems_remaining; j++, quad_data_cursor=quad_data_cursor + quad_data_size) {
      iter_volume(
        p4est,
        ghost_layer,
        quad_cursor++,
        quad_data_cursor,
        cursor++,
        treeId,
        user_data
      );
    }
  }
}

void run_setup_kernel_volume_callback(cuda_iter_volume_api_t* iter_volume_api, cuda_iter_volume_t* d_callback) {
    iter_volume_api->setup_kernel<<<1,1>>>(d_callback);
}

void run_setup_kernel_face_callback(cuda_iter_face_api_t* iter_face_api, cuda_iter_face_t* d_callback) {
    iter_face_api->setup_kernel<<<1,1>>>(d_callback);
}

void run_setup_new_kernel_face_callback(cuda_new_iter_face_api_t* new_iter_face_api, cuda_new_iter_face_t* d_callback) {
  new_iter_face_api->setup_kernel<<<1,1>>>(d_callback);
}

void run_simple_quadrants_iterate(p4est_quadrant_t* quadrants, void* quads_data, size_t quad_data_size,
    p4est_ghost_t* ghost_layer,
    p4est_t* p4est, p4est_topidx_t treeId, 
    void* user_data, cuda_iter_volume_t iter_volume,
    size_t quads_count, size_t quads_per_thread,
    size_t needed_block_count, size_t threads_per_block
) {
    simple_quadrants_iterate<<<needed_block_count, threads_per_block>>>(
        quadrants, quads_data, quad_data_size,
        ghost_layer,
        p4est, treeId,
        user_data, iter_volume,
        quads_count, quads_per_thread
    );
    gpuErrchk(hipDeviceSynchronize());
}

__global__ void
simple_faces_iterate(
    p4est_t* p4est, p4est_ghost_t* ghost_layer,
    sc_array_t* quadrants, 
    p4est_iter_face_side_t* faces, size_t faces_count,
    void* user_data, cuda_iter_face_t iter_face,
    size_t faces_per_iter, size_t faces_per_thread)
{
//  sc_array_t *ghost_quadrants = &(ghost_layer->ghosts);
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < faces_count) {
    size_t elems_remaining;
    if(i >= faces_count - faces_per_thread) {
      elems_remaining = faces_count - i;
    } else {
      elems_remaining = faces_per_thread;
    }
    p4est_iter_face_side_t* cursor = faces + i * faces_per_thread * faces_per_iter;
    for(int j = 0; j < elems_remaining; j++, cursor+=faces_per_iter) {
        iter_face(
            p4est,
            ghost_layer,
            cursor,
            user_data
        );
    }
  }
}

void run_simple_faces_iterate(p4est_t* p4est, p4est_ghost_t* ghost_layer,
    sc_array_t* quadrants,
    p4est_iter_face_side_t* faces, size_t faces_count,
    void* user_data, cuda_iter_face_t iter_face,
    size_t faces_per_iter, size_t faces_per_thread, size_t needed_block_count, size_t threads_per_block) {
    
    simple_faces_iterate<<<needed_block_count, threads_per_block>>>(
        p4est, ghost_layer,
        quadrants,
        faces, faces_count,
        user_data, iter_face,
        faces_per_iter, faces_per_thread
    );
    gpuErrchk(hipDeviceSynchronize());
}

__global__ void
simple_new_faces_iterate(
    p4est_t* p4est,
    size_t *block_configs,
    void *blocks_user_data,
    unsigned char* quads_levels,
    cuda_light_face_side_t* sides,
    void* user_data, cuda_new_iter_face_t new_iter_face)
{ 
  extern __shared__ char array[];

  size_t *global_block_config = block_configs + blockIdx.x * 7;

  size_t global_block_quads_start_index = global_block_config[0];
  size_t global_block_quads_count = global_block_config[1];

  size_t global_block_start_byte_index = global_block_config[2];
  size_t global_block_quads_bytes_count = global_block_config[3];
  
  size_t global_block_output_quads_count = global_block_config[4];

  size_t global_block_faces_start_index = global_block_config[5];
  size_t global_block_faces_count = global_block_config[6];
  
  
  char* block_user_data = (char*)array;
  unsigned char* block_quads_levels = (unsigned char*)(global_block_quads_bytes_count + array); 
  cuda_light_face_side_t *face_sides = (cuda_light_face_side_t*)(global_block_quads_bytes_count + global_block_quads_count + array);

  /*
  if(threadIdx.x == 0){
    printf("[cuda] %d-block  %lu, %lu, %lu, %lu, %lu, %lu, %lu\n",
      blockIdx.x,
      global_block_quads_start_index,
      global_block_quads_count,
      global_block_start_byte_index,
      global_block_quads_bytes_count,
      global_block_output_quads_count,
      global_block_faces_start_index,
      global_block_faces_count
    );
  }
  */
  
  
  size_t faces_count = global_block_faces_count;
  size_t faces_per_thread;
  if(faces_count % blockDim.x) {
    faces_per_thread = faces_count / blockDim.x + 1;
  } else {
    faces_per_thread = faces_count / blockDim.x;
  }
  
  size_t quads_count = global_block_quads_count;
  size_t quads_per_thread;
  if(quads_count % blockDim.x) { 
    quads_per_thread = quads_count / blockDim.x + 1;
  } else {
    quads_per_thread = quads_count / blockDim.x;
  }

  /*
  if(threadIdx.x == 0){
    printf("[cuda] %d-block faces_count: %lu, faces_per_thread: %lu\n",
      blockIdx.x,
      faces_count,
      faces_per_thread
    );
  }
  */

  /*
  if(threadIdx.x == 0 && blockIdx.x == 0) {
    printf("0 - memory_size: %d\n", global_block_quads_bytes_count + global_block_quads_count + faces_count * sizeof(cuda_light_face_side_t) * 2);
  }

  if(threadIdx.x == 0 && blockIdx.x == 1) {
    printf("1 - memory_size: %d\n", global_block_quads_bytes_count + global_block_quads_count + faces_count * sizeof(cuda_light_face_side_t) * 2);
  }
  */
  
  
  
  int i = threadIdx.x;
  size_t faces_remaining = 0;
  size_t face_start_index = faces_per_thread * i;
  if(face_start_index < faces_count){
    faces_remaining = faces_count - face_start_index;
    if(faces_remaining > faces_per_thread) {
      faces_remaining = faces_per_thread;
    }
  }
  size_t sides_start_index = (global_block_faces_start_index + face_start_index) * 2;
  size_t local_start_index = face_start_index * 2;

  for(size_t j = 0; j < faces_remaining; j++, sides_start_index++, local_start_index++) {
    face_sides[local_start_index] = sides[sides_start_index];
    sides_start_index++;
    local_start_index++;
    face_sides[local_start_index] = sides[sides_start_index];
  }
  
  size_t quads_remaining = 0;
  size_t quad_start_index = quads_per_thread * i;
  if(quad_start_index < quads_count){
    quads_remaining = quads_count - quad_start_index;
    if(quads_remaining > quads_per_thread) {
      quads_remaining = quads_per_thread;
    }
  }
  
  size_t quad_start_byte = global_block_faces_start_index;
  size_t user_data_size = global_block_quads_bytes_count / global_block_quads_count;

  for(size_t j = 0; j < quads_remaining; j++, quad_start_byte+=user_data_size) {
    size_t user_data_byte_end = quad_start_byte + user_data_size;
    for(size_t byte_index = quad_start_byte; byte_index < user_data_byte_end; byte_index++) {
      block_user_data[byte_index] = (char)(blocks_user_data + byte_index);
    }
    size_t current_quad_index = quad_start_index + j;
    block_quads_levels[current_quad_index] = quads_levels[current_quad_index];
  }
  __shared__ size_t output_quads_count;
  output_quads_count = global_block_output_quads_count;
  
  __syncthreads();

  size_t face_side_start_index = face_start_index * 2;
  size_t sides_remaining = faces_remaining * 2;
  for(size_t j = 0; j < sides_remaining; j+=2) {
    new_iter_face(
      p4est,
      output_quads_count,
      block_user_data,
      block_quads_levels,
      face_sides + face_side_start_index + j
    );
  }
}

void run_new_simple_faces_iterate(p4est_t* p4est,
  size_t block_count,
  size_t *block_configs,
  void *blocks_user_data,
  unsigned char* quads_levels,
  cuda_light_face_side_t* sides,
  size_t shared_memory_size,
  void* user_data, cuda_new_iter_face_t new_iter_face
){
  simple_new_faces_iterate<<<block_count,128, shared_memory_size>>>(p4est, block_configs, blocks_user_data, quads_levels, sides, user_data, new_iter_face);
  gpuErrchk(hipDeviceSynchronize());
}
