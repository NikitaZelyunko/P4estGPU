#include "hip/hip_runtime.h"
#include "simple_cuda_iterate.h"

__global__ void
simple_quadrants_iterate(
  sc_array_t* quadrants, p4est_ghost_t* ghost_layer,
  p4est_t* p4est, p4est_topidx_t treeId, 
  void* user_data, cuda_iter_volume_t iter_volume,
  size_t quads_count, size_t quads_per_thread)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < quads_count) {
    size_t elems_remaining;
    if(i >= quads_count - quads_per_thread) {
      elems_remaining = quads_count - i;
    } else {
      elems_remaining = quads_per_thread;
    }
    for(int j = 0; j < elems_remaining; j++) {
      iter_volume(
        p4est,
        ghost_layer,
        p4est_device_quadrant_array_index(quadrants, i + j),
        i+j,
        treeId,
        user_data
      );
    }
  }
}

void run_setup_kernel_volume_callback(cuda_iter_volume_api_t* iter_volume_api, cuda_iter_volume_t* d_callback) {
    iter_volume_api->setup_kernel<<<1,1>>>(d_callback);
}

void run_setup_kernel_face_callback(cuda_iter_face_api_t* iter_face_api, cuda_iter_face_t* d_callback) {
    iter_face_api->setup_kernel<<<1,1>>>(d_callback);
}

void run_simple_quadrants_iterate(sc_array_t* quadrants, p4est_ghost_t* ghost_layer,
    p4est_t* p4est, p4est_topidx_t treeId, 
    void* user_data, cuda_iter_volume_t iter_volume,
    size_t quads_count, size_t quads_per_thread,
    size_t needed_block_count, size_t threads_per_block
) {

    simple_quadrants_iterate<<<needed_block_count, threads_per_block>>>(
        quadrants, ghost_layer,
        p4est, treeId,
        user_data, iter_volume,
        quads_count, quads_per_thread
    );
    gpuErrchk(hipDeviceSynchronize());
}

__global__ void
simple_faces_iterate(
    p4est_t* p4est, p4est_ghost_t* ghost_layer,
    sc_array_t* quadrants, 
    p4est_iter_face_side_t* faces, size_t faces_count,
    void* user_data, cuda_iter_face_t iter_face,
    size_t faces_per_iter, size_t faces_per_thread)
{
  sc_array_t *ghost_quadrants = &(ghost_layer->ghosts);
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < faces_count) {
    size_t elems_remaining;
    if(i >= faces_count - faces_per_thread) {
      elems_remaining = faces_count - i;
    } else {
      elems_remaining = faces_per_thread;
    }
    p4est_iter_face_side_t* cursor = faces + i * faces_per_iter;
    for(int j = 0; j < elems_remaining; j++, cursor+=faces_per_iter) {
        p4est_iter_face_side_t* current_face = cursor;
        if(current_face->is_hanging) {
          p4est_quadrant_t** quads = current_face->is.hanging.quad;
          p4est_locidx_t *quadid = current_face->is.hanging.quadid;
          if(current_face->is.hanging.is_ghost[0]){
            quads[0] = p4est_device_quadrant_array_index(ghost_quadrants, quadid[0]);
          } else {
            quads[0] = p4est_device_quadrant_array_index(quadrants, quadid[0]);
          }

          if(current_face->is.hanging.is_ghost[1]) {
            quads[1] = p4est_device_quadrant_array_index(ghost_quadrants, quadid[1]);
          } else {
            quads[1] = p4est_device_quadrant_array_index(quadrants, quadid[1]);
          }
          
        } else {
          if(current_face->is.full.is_ghost){
            current_face->is.full.quad = p4est_device_quadrant_array_index(ghost_quadrants, current_face->is.full.quadid);
          } else {
            current_face->is.full.quad = p4est_device_quadrant_array_index(quadrants, current_face->is.full.quadid);
          }
        }
        current_face++;
        if(current_face->is_hanging) {
          p4est_quadrant_t** quads = current_face->is.hanging.quad;
          p4est_locidx_t *quadid = current_face->is.hanging.quadid;
          
          if(current_face->is.hanging.is_ghost[0]){
            quads[0] = p4est_device_quadrant_array_index(ghost_quadrants, quadid[0]);
          } else {
            quads[0] = p4est_device_quadrant_array_index(quadrants, quadid[0]);
          }

          if(current_face->is.hanging.is_ghost[1]) {
            quads[1] = p4est_device_quadrant_array_index(ghost_quadrants, quadid[1]);
          } else {
            quads[1] = p4est_device_quadrant_array_index(quadrants, quadid[1]);
          }
        } else {
          if(current_face->is.full.is_ghost){
            current_face->is.full.quad = p4est_device_quadrant_array_index(ghost_quadrants, current_face->is.full.quadid);
          } else {
            current_face->is.full.quad = p4est_device_quadrant_array_index(quadrants, current_face->is.full.quadid);
          }
        }
        iter_face(
            p4est,
            ghost_layer,
            cursor,
            user_data
        );
    }
  }
}

void run_simple_faces_iterate(p4est_t* p4est, p4est_ghost_t* ghost_layer,
    sc_array_t* quadrants,
    p4est_iter_face_side_t* faces, size_t faces_count,
    void* user_data, cuda_iter_face_t iter_face,
    size_t faces_per_iter, size_t faces_per_thread, size_t needed_block_count, size_t threads_per_block) {
    
    simple_faces_iterate<<<needed_block_count, threads_per_block>>>(
        p4est, ghost_layer,
        quadrants,
        faces, faces_count,
        user_data, iter_face,
        faces_per_iter, faces_per_thread
    );
    gpuErrchk(hipDeviceSynchronize());
}
